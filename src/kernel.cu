#include "hip/hip_runtime.h"
#include "Particle.h"
#include "kernel.cuh"

//calculate force points
__global__ void PointsForce(int particlesN, int maxPointsN, float pointMassMultiplier, float* posX, float* posY, float* fx, float* fy, float* pointX, float* pointY, float* pointM)
{
	int i = threadIdx.x * blockIdx.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < particlesN; i += stride)
	{
		float dist;

		for (int j = 0; j < maxPointsN; j++)
		{
			if (pointM[j] != 0.0f)
			{
				dist = sqrtf(pow(posX[i] - pointX[j], 2) + pow(posY[i] - pointY[j], 2));

				fx[i] += pointMassMultiplier * 0.1 * pointM[j] * (posX[i] - pointX[j]) / pow(dist, 3.0);
				fy[i] += pointMassMultiplier * 0.1 * pointM[j] * (posY[i] - pointY[j]) / pow(dist, 3.0);
			}
		}
	}
}

//calculate force between particles
__global__ void ParticlesForce(int particlesN, float particleMassMultiplier, float* posX, float* posY, float* m, float* fx, float* fy)
{
	int i = threadIdx.x * blockIdx.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < particlesN; i += stride)
	{
		float dist;

		for (int j = 0; j < particlesN; j++)
		{
			if (i != j)
			{
				dist = sqrtf(pow(posX[i] - posX[j], 2) + pow(posY[i] - posY[j], 2));
				fx[i] += particleMassMultiplier * 0.1 * m[j] * (posX[i] - posX[j]) / pow(dist, 3.0);
				fy[i] += particleMassMultiplier * 0.1 * m[j] * (posY[i] - posY[j]) / pow(dist, 3.0);
			}

		}
	}
}

//Update particles
__global__ void UpdateParticlesPosition(int particlesN, float particleMassMultiplier, float pointMassMultiplier, float viscosity1, float viscosity2, float dt, float* posX, float* posY, float* velX, float* velY, float* accX, float* accY, float* fx, float* fy, float* m, float* r, float* pointX, float* pointY, float* pointM)
{
	int i = threadIdx.x * blockIdx.x;
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < particlesN; i += stride)
	{
		float addfx;
		float addfy;
		float dx = 0;
		float dy = 0;
		float q1 = viscosity1;
		float q2 = viscosity2;
		float dist;

		//apply velocity
		velX[i] = velX[i] + accX[i] * dt;
		velY[i] = velY[i] + accY[i] * dt;

		//max vel
		float max = 100;

		if (velX[i] > max) velX[i] = max;
		if (velX[i] < -max) velX[i] = -max;
		if (velY[i] > max) velY[i] = max;
		if (velY[i] < -max) velY[i] = -max;

		//apply possition
		posX[i] = posX[i] + velX[i] * dt;
		posY[i] = posY[i] + velY[i] * dt;

		//apply aerodynamic drag
		float gr = 0.1;
		if (velX[i] > gr || velX[i] < -gr || velY[i] > gr || velY[i] < -gr)
		{
			if (pow(posX[i] - 512.0, 2) + pow(posY[i] - 512.0, 2) >= pow(400.0, 2))
			{
				dx = -6 * 3.14 * velX[i] * q2 * r[i];
				dy = -6 * 3.14 * velY[i] * q2 * r[i];
			}
			else
			{
				dx = -6 * 3.14 * velX[i] * q1 * r[i];
				dy = -6 * 3.14 * velY[i] * q1 * r[i];
			}
		}
		else
		{
			velX[i] = 0.0;
			velY[i] = 0.0;
		}

		fx[i] -= dx;
		fy[i] -= dy;

		//apply acceleration
		accX[i] = -fx[i] / m[i];
		accY[i] = -fy[i] / m[i];

		//apply boundaries
		if (posX[i] - r[i] < 0) velX[i] = -velX[i];
		if (posX[i] + r[i] > 1024) velX[i] = -velX[i];

		if (posY[i] - r[i] < 0) velY[i] = -velY[i];
		if (posY[i] + r[i] > 1024) velY[i] = -velY[i];

		fx[i] = 0.0;
		fy[i] = 0.0;
	}
}

Cuda::Cuda(int cudaParticlesN, int cudaPointsN, int cudaMaxPointsN, vector<Particle>* particles, vector<Particle*>* points)
{
	particlesN = cudaParticlesN;
	pointsN = cudaPointsN;
	maxPointsN = cudaMaxPointsN;

	int* idHost = new int[particlesN];
	float* rHost = new float[particlesN];
	float* mHost = new float[particlesN];
	float* posXHost = new float[particlesN];
	float* posYHost = new float[particlesN];
	float* velXHost = new float[particlesN];
	float* velYHost = new float[particlesN];
	float* accXHost = new float[particlesN];
	float* accYHost = new float[particlesN];
	float* distHost = new float[particlesN];
	float* rayXHost = new float[particlesN];
	float* rayYHost = new float[particlesN];
	float* pointXHost = new float[maxPointsN];
	float* pointYHost = new float[maxPointsN];
	float* pointMHost = new float[maxPointsN];

	hipMalloc(&id, particlesN * sizeof(int));
	hipMalloc(&r, particlesN * sizeof(float));
	hipMalloc(&m, particlesN * sizeof(float));
	hipMalloc(&posX, particlesN * sizeof(float));
	hipMalloc(&posY, particlesN * sizeof(float));
	hipMalloc(&velX, particlesN * sizeof(float));
	hipMalloc(&velY, particlesN * sizeof(float));
	hipMalloc(&accX, particlesN * sizeof(float));
	hipMalloc(&accY, particlesN * sizeof(float));
	hipMalloc(&fx, particlesN * sizeof(float));
	hipMalloc(&fy, particlesN * sizeof(float));
	hipMalloc(&dist, particlesN * sizeof(float));
	hipMalloc(&rayX, particlesN * sizeof(float));
	hipMalloc(&rayY, particlesN * sizeof(float));
	hipMalloc(&pointX, maxPointsN * sizeof(float));
	hipMalloc(&pointY, maxPointsN * sizeof(float));
	hipMalloc(&pointM, maxPointsN * sizeof(float));

	for (int i = 0; i < particlesN; i++)
	{
		idHost[i] = particles->at(i).getID();
		rHost[i] = particles->at(i).getRadius();
		mHost[i] = particles->at(i).getMass();
		posXHost[i] = particles->at(i).getPosition().x;
		posYHost[i] = particles->at(i).getPosition().y;
		velXHost[i] = particles->at(i).getVelocity().x;
		velYHost[i] = particles->at(i).getVelocity().y;
		accXHost[i] = particles->at(i).getAcceleration().x;
		accYHost[i] = particles->at(i).getAcceleration().y;
	}

	for (int i = 0; i < maxPointsN; i++)
	{
		pointXHost[i] = 0.0f;
		pointYHost[i] = 0.0f;
		pointMHost[i] = 0.0f;
	}

	for (int i = 0; i < pointsN; i++)
	{
		pointXHost[i] = points->at(i)->getPosition().x;
		pointYHost[i] = points->at(i)->getPosition().y;
		pointMHost[i] = points->at(i)->getMass();
	}

	hipMemcpy(r, rHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(m, mHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(posX, posXHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(posY, posYHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(velX, velXHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(velY, velYHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(accX, accXHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(accY, accYHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dist, distHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(rayX, rayXHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(rayY, rayYHost, particlesN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pointX, pointXHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pointY, pointYHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pointM, pointMHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);

	free(rHost);
	free(mHost);
	free(posXHost);
	free(posYHost);
	free(velXHost);
	free(velYHost);
	free(accXHost);
	free(accYHost);
	free(distHost);
	free(rayXHost);
	free(rayYHost);
	free(pointXHost);
	free(pointYHost);
	free(pointMHost);
}

Cuda::~Cuda()
{

}

void Cuda::run(vector<Particle>* particles, float particleMassMultiplier, float pointMassMultiplier, float viscosity1, float viscosity2, float dt)
{
	int blockSize = 128;
	int numBlocks = (particlesN + blockSize - 1) / blockSize;
	PointsForce << < numBlocks, blockSize >> > (particlesN, maxPointsN, pointMassMultiplier, posX, posY, fx, fy, pointX, pointY, pointM);
	hipDeviceSynchronize();
	ParticlesForce << < numBlocks, blockSize >> > (particlesN, particleMassMultiplier, posX, posY, m, fx, fy);
	hipDeviceSynchronize();
	//Collision << < numBlocks, blockSize >> > (particlesN, pointsN, posX, posY, velX, velY, r, m, pointX, pointY);
	//hipDeviceSynchronize();
	UpdateParticlesPosition << < numBlocks, blockSize >> > (particlesN, particleMassMultiplier, pointMassMultiplier, viscosity1, viscosity2, dt, posX, posY, velX, velY, accX, accY, fx, fy, m, r, pointX, pointY, pointM);
	hipDeviceSynchronize();

	float* posXHost = new float[particlesN];
	float* posYHost = new float[particlesN];
	float* velXHost = new float[particlesN];
	float* velYHost = new float[particlesN];
	float* accXHost = new float[particlesN];
	float* accYHost = new float[particlesN];

	hipMemcpy(posXHost, posX, particlesN * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(posYHost, posY, particlesN * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(velXHost, velX, particlesN * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(velYHost, velY, particlesN * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(accXHost, accX, particlesN * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(accYHost, accY, particlesN * sizeof(float), hipMemcpyDeviceToHost);


	for (int i = 0; i < particlesN; i++)
	{
		particles->at(i).setPosition(v2(posXHost[i], posYHost[i]));
		particles->at(i).setVelocity(v2(velXHost[i], velYHost[i]));
		particles->at(i).setAcceleration(v2(accXHost[i], accYHost[i]));
	}

	free(posXHost);
	free(posYHost);
	free(velXHost);
	free(velYHost);
	free(accXHost);
	free(accYHost);
}

void Cuda::UpdatePoint(vector<Particle*>* points)
{
	pointsN = points->size();

	float* pointXHost = new float[maxPointsN];
	float* pointYHost = new float[maxPointsN];
	float* pointMHost = new float[maxPointsN];

	for (int i = 0; i < maxPointsN; i++)
	{
		pointXHost[i] = 0.0f;
		pointYHost[i] = 0.0f;
		pointMHost[i] = 0.0f;
	}

	for (int j = 0; j < pointsN; j++)
	{
		pointXHost[j] = points->at(j)->getPosition().x;
		pointYHost[j] = points->at(j)->getPosition().y;
		pointMHost[j] = points->at(j)->getMass();
	}

	int blockSize = 256;
	int numBlocks = (particlesN + blockSize - 1) / blockSize;

	hipMemcpy(pointX, pointXHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pointY, pointYHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pointM, pointMHost, maxPointsN * sizeof(float), hipMemcpyHostToDevice);

	free(pointXHost);
	free(pointYHost);
	free(pointMHost);
}
